
#include <hip/hip_runtime.h>
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 * 
 *   http://www.apache.org/licenses/LICENSE-2.0
 * 
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */
 
/**********************************
When updating a kernel or adding a new one, 
please compile the ptx file and commit it:
nvcc -ptx SystemML.cu 
***********************************/

// dim => rlen (Assumption: rlen == clen)
// N = length of dense array
extern "C"
__global__ void copyUpperToLowerTriangleDense(double* ret, int dim, int N) {
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;
	int id_dest = iy * dim + ix;
	if(iy > ix && id_dest < N) {
		// TODO: Potential to reduce the number of threads by half
		int id_src = ix * dim + iy;
		ret[id_dest] = ret[id_src];
	}
}