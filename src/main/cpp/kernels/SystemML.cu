
#include <hip/hip_runtime.h>
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 * 
 *   http://www.apache.org/licenses/LICENSE-2.0
 * 
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */
 
/**********************************
When updating a kernel or adding a new one, 
please compile the ptx file and commit it:
nvcc -ptx SystemML.cu 
***********************************/

// dim => rlen (Assumption: rlen == clen)
// N = length of dense array
extern "C"
__global__ void copyUpperToLowerTriangleDense(double* ret, int dim, int N) {
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;
	int id_dest = iy * dim + ix;
	if(iy > ix && id_dest < N) {
		// TODO: Potential to reduce the number of threads by half
		int id_src = ix * dim + iy;
		ret[id_dest] = ret[id_src];
	}
}

extern "C"
__device__ double getBoolean(int val) {
	if(val == 0)
		return 0.0;
	else
		return 1.0;
}

// op = {0=plus, 1=minus, 2=multiply, 3=divide, 4=power, 
// 5=less, 6=lessequal, 7=greater, 8=greaterequal, 9=equal, 10=notequal, 
// 11=min, 12=max, 13=and, 14=or, 15=log}
extern "C"
__device__ double binaryOp(double x, double y, int op) {
	// 0=plus, 1=minus, 2=multiply, 3=divide, 4=power
	if(op == 0)
		return x + y;
	else if(op == 1)
		return x - y;
	else if(op == 2)
		return x * y;
	else if(op == 3)
		return x / y;
	else if(op == 4)
		return pow(x, y);
	// 5=less, 6=lessequal, 7=greater, 8=greaterequal, 9=equal, 10=notequal,	
	else if(op == 5) 
		return getBoolean(x < y);
	else if(op == 6)
		return getBoolean(x <= y);
	else if(op == 7)
		return getBoolean(x > y);
	else if(op == 8)
		return getBoolean(x >= y);
	else if(op == 9)
		return getBoolean(x == y);
	else if(op == 10)
		return getBoolean(x != y);
	// 11=min, 12=max, 13=and, 14=or, 15=log
	else if(op == 11) {
		return min(x, y);
	}
	else if(op == 12) {
		return max(x, y);
	}
	return -999;
}

extern "C"
__global__ void dense_matrix_set(double* A,  double scalar, int rlen, int clen) {
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;
	int index = ix * clen + iy;
	if(index < rlen*clen) {
		A[index] = scalar;
	}	
}

extern "C"
__global__ void dense_matrix_copy(double* A,  double* ret, int rlen, int clen) {
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;
	int index = ix * clen + iy;
	if(ix < rlen && iy < clen) {
		ret[index] = A[index];
	}
}

// Compares the value and set
extern "C"
__global__ void compareAndSet(double* A,  double* ret, int rlen, int clen, double compareVal, double tol, double ifEqualsVal, double ifLessThanVal, double ifGreaterThanVal) {
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;
	int index = ix * clen + iy;
	if(ix < rlen && iy < clen) {
		if(abs(A[index]-compareVal) < tol)
			ret[index] = ifEqualsVal;
		else if(A[index] < compareVal)
			ret[index] = ifLessThanVal;
		else		
			ret[index] = ifGreaterThanVal;
	}
}

extern "C"
__global__ void binCellOp(double* A, double* B, double* C, 
	int maxRlen, int maxClen, int vectorAStatus, int vectorBStatus, int op) {
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;
	
	if(ix < maxRlen && iy < maxClen) {
		int outIndex = ix * maxClen + iy;
		int aIndex = outIndex;
		int bIndex = outIndex;
		if(vectorAStatus == 1)
			aIndex = ix; // clen == 1
		else if(vectorAStatus == 2)
			aIndex = iy; // rlen == 1
		if(vectorBStatus == 1)
			bIndex = ix; // clen == 1
		else if(vectorBStatus == 2)
			bIndex = iy; // rlen == 1
		C[outIndex] = binaryOp(A[aIndex], B[bIndex], op);
		// printf("C[%d] = A[%d](%f) B[%d](%f) (%d %d)\n", outIndex, aIndex, A[aIndex], bIndex,  B[bIndex], (ix+1), (iy+1));
	}
}

extern "C"
__global__ void binCellScalarOp(double* A, double scalar, double* C, int rlenA, int clenA, int op, int isLeftScalar) {
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;
	int index = ix * clenA + iy;
	if(index < rlenA*clenA) {
		if(isLeftScalar)
			C[index] = binaryOp(scalar, A[index], op);
		else
			C[index] = binaryOp(A[index], scalar, op);
	}
}